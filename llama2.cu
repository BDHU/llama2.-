#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <stdint.h>
#include <assert.h>

#include "hip/hip_runtime.h"


#ifndef checkCudaErrors
#define checkCudaErrors(err) __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if (hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}
#endif

// ----------------------------------------------------------------------------
// The Byte Pair Encoding (BPE) Tokenizer that translates strings <-> tokens
// ----------------------------------------------------------------------------

typedef struct {
    char *str;
    int id;
} TokenIndex;

typedef struct {
    char **vocab;
    float *vocab_scores;
    TokenIndex *sorted_vocab;
    int vocab_size;
    unsigned int max_token_length;
    unsigned char byte_pieces[512]; // stores all single-byte strings
} Tokenizer;

// ----------------------------------------------------------------------------
// The Sampler, which takes logits and returns a sampled token
// sampling can be done in a few ways: greedy argmax, sampling, top-p sampling
// ----------------------------------------------------------------------------

typedef struct {
    float prob;
    int index;
} ProbIndex; // struct used when sorting probabilities during top-p sampling

typedef struct {
    int vocab_size;
    ProbIndex *probindex;   // buffer used in top-p sampling
    float temperature;
    float topp;
    unsigned long long rng_state;
} Sampler;


// ----------------------------------------------------------------------------
// Transformer-related structs
// ----------------------------------------------------------------------------

typedef struct {
    int dim; // transformer dimension
    int hidden_dim; // ffn layer dimension
    int n_layers; // number of transformer layers
    int n_heads; // number of query heads
    int n_kv_heads; // number of k/v heads
    int vocab_size; // vocabulary size, usually 256 (byte-level)
    int max_seq_len; // maximum sequence length to generate
} Config;

typedef struct {
    // token embedding table
    float* token_embedding_table;    // (vocab_size, dim)
    // weights for rmsnorms
    float* rms_att_weight; // (layer, dim) rmsnorm weights
    float* rms_ffn_weight; // (layer, dim)
    // weights for matmuls. note dim == n_heads * head_size
    float* wq; // (layer, dim, n_heads * head_size)
    float* wk; // (layer, dim, n_kv_heads * head_size)
    float* wv; // (layer, dim, n_kv_heads * head_size)
    float* wo; // (layer, n_heads * head_size, dim)
    // weights for ffn
    float* w1; // (layer, hidden_dim, dim)
    float* w2; // (layer, dim, hidden_dim)
    float* w3; // (layer, hidden_dim, dim)
    // final rmsnorm
    float* rms_final_weight; // (dim,)
    // (optional) classifier weights for the logits, on the last layer
    float* wcls;
} TransformerWeights;

// RunState definition
typedef struct {
    float *x; // activation at current time stamp (dim,)
    float *xb; // same, but insize a residual branch (dim,)
    float *xb2; // an additional buffer just for convenience (dim,)
    float *hb; // buffer for hidden dimension in the ffn (hidden_dim,)
    float *hb2; // bufffer for hidden dimension in the ffn (hidden_dim,)
    float *q; // query (dim,)
    float *k; // key (dim,)
    float *v; // value (dim,)
    float *att; // buffer for the scores/attention avlues (n_heads, seq_len)
    float *logits; // output logits
    // kv cache
    float *key_cache; // (layer, seq_len, dim)
    float *value_cache; // (layer , seq_len, dim)
} RunState;

// Transformer definition
typedef struct {
    Config config;
    TransformerWeights weights; // model weights
    RunState state; // buffer required to store intermediate values during forward pass
    int fd; // file descriptor required for memory mapping, explained later TODO
    float *data; // data pointer, TODO
    uint64_t file_size; // size of the model checkpoint file in bytes
} Transformer;

// ----------------------------------------------------------------------------
// Sampler
// ----------------------------------------------------------------------------

void build_sampler(Sampler* sampler, int vocab_size, float temperature, float topp, unsigned long long rng_seed) {
    sampler->vocab_size = vocab_size;
    sampler->temperature = temperature;
    sampler->topp = topp;
    sampler->rng_state = rng_seed;
    // buffer only used with nucleus sampling; may not need but it's ~small
    sampler->probindex = (ProbIndex *)malloc(sampler->vocab_size * sizeof(ProbIndex));
}

void free_sampler(Sampler *sampler) {
    free(sampler->probindex);
}

// ----------------------------------------------------------------------------
// Tokenizer
// ----------------------------------------------------------------------------

int compare_tokens(const void *a, const void *b) {
    return strcmp(((TokenIndex*)a)->str, ((TokenIndex*)b)->str);
}

void build_tokenizer(Tokenizer *t, char *tokenizer_path, int vocab_size) {
    // should've written the vocab_size into the tokenizer file... sigh
    t->vocab_size = vocab_size;
    // allocate space to hold the scores and the strings
    t->vocab = (char **)malloc(vocab_size * sizeof(char *));
    t->vocab_scores = (float *)malloc(vocab_size * sizeof(float));
    t->sorted_vocab = NULL; // initialized lazily
    for (int i = 0; i < 256; i++) {
        t->byte_pieces[i * 2] = (unsigned char)i;
        t->byte_pieces[i * 2 + 1] = '\0';
    }
    // read in the file
    FILE *file = fopen(tokenizer_path, "rb");
    if (!file) {fprintf(stderr, "couldn't load %s\n", tokenizer_path); exit(EXIT_FAILURE);}
    if (fread(&t->max_token_length, sizeof(int), 1, file) != 1) {fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE);}
    int len = 0;
    for (int i = 0; i < vocab_size; i++) {
        if (fread(t->vocab_scores + i, sizeof(float), 1, file) != 1) {fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE);}
        if (fread(&len, sizeof(int), 1, file) != 1) {fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE);}
        t->vocab[i] = (char *)malloc(len + 1);
        if (fread(t->vocab[i], len, 1, file) != 1) {fprintf(stderr, "failed read\n"); exit(EXIT_FAILURE);}
        t->vocab[i][len] = '\0'; // add the string terminating token
    }
    fclose(file);
}

void free_tokenizer(Tokenizer *t) {
    for (int i = 0; i < t->vocab_size; i++) { free(t->vocab[i]); }
    free(t->vocab);
    free(t->vocab_scores);
    free(t->sorted_vocab);
}

int str_lookup(char *str, const TokenIndex *sorted_vocab, size_t vocab_size) {
    // efficiently find the perfect match for str in vocab, return its index or -1 if not found
    const TokenIndex tok = { .str = str }; // acts as the key to search for
    TokenIndex *res = (TokenIndex *)bsearch((const void *)&tok, sorted_vocab, vocab_size, sizeof(TokenIndex), compare_tokens);
    return res != NULL ? res->id : -1;
}

void encode(Tokenizer *t, char *text, int8_t bos, int8_t eos, int *tokens, int *n_tokens) {
    // encode the string text (input) into an upper-bound preallocated tokens[] array
    // bos != 0 means prepend the BOS token (=1), eos != 0 means append the EOS token (=2)
    if (text == NULL) {fprintf(stderr, "cannot encode NULL text\n"); exit(EXIT_FAILURE);}

    printf("vocab_size is %d\n", t->vocab_size);
    if (t->sorted_vocab == NULL) {
        // lazily alloc and sort the vocabulary
        checkCudaErrors(hipMallocManaged((void **)&t->sorted_vocab, t->vocab_size * sizeof(TokenIndex)));
        for (int i = 0; i < t->vocab_size; i++) {
            t->sorted_vocab[i].str = t->vocab[i];
            t->sorted_vocab[i].id = i;
        }
        qsort(t->sorted_vocab, t->vocab_size, sizeof(TokenIndex), compare_tokens);
    }

    // create a temporary buffer that will store merge candidates of always two consecutive tokens
    // *2 for concat, +1 for null terminator +2 for UTF8 (in case max_token_length is 1)
    char *str_buffer = (char *)malloc((t->max_token_length*2 +1 +2) * sizeof(char));
    printf("max token len: %d\n", t->max_token_length);
    size_t str_len = 0;

    // start at 0 tokens
    *n_tokens = 0;

    // add optional BOS (=1) token, if desired
    if (bos) {tokens[(*n_tokens)++] = 1;}

    // add_dummy_prefix is true by default
    // so prepend a dummy prefix token to the input string, but only if text != ""
    // TODO: pretty sure this isn't correct in the general case but I don't have the
    // energy to read more of the sentencepiece code to figure out what it's doing
    if (text[0] != '\0') {
        int dummy_prefix = str_lookup(" ", t->sorted_vocab, t->vocab_size);
        printf("dummy prefix %d\n", dummy_prefix);
        tokens[(*n_tokens)++] = dummy_prefix;
    }

    // Okay UTF-8 time. This will get messy. Here is the reference from Wikipedia:
    // Code point ↔ UTF-8 conversion
    // First code point	Last code point	Byte 1	Byte 2	Byte 3	Byte 4
    // U+0000	U+007F	    0xxxxxxx
    // U+0080	U+07FF	    110xxxxx	10xxxxxx
    // U+0800	U+FFFF	    1110xxxx	10xxxxxx	10xxxxxx
    // U+10000	U+10FFFF    11110xxx	10xxxxxx	10xxxxxx	10xxxxxx

    // process the raw (UTF-8) byte sequence of the input string
    const int max_num_utf8_byte = 4;
    for (char *c = text; *c != '\0'; c++) {
        // reset buffer if the current byte is ASCII or a leading byte
        // 0xC0 is 11000000, so (*c & 0xC0) keeps the first 2 bits and zeros the rest
        // 0x80 is 10000000
        // in UTF-8, all continuation bytes start with "10" in first two bits
        // so in English this is: "if this byte is not a continuation byte"
        if ((*c & 0xC0) != 0x80) {
            // this byte must be either a leading byte (11...) or an ASCII char (0x...)
            // => reset our location, as we're starting a new UTF-8 codepoint
            str_len = 0;
        }

        // append the current byte to the buffer
        str_buffer[str_len++] = *c;
        str_buffer[str_len] = '\0'; // write null char in case we have an ASCII character

        // while the next character is a continuation byte, continue appending
        // but if there are too many of them, just stop to avoid overruning str_buffer size.
        if ((*(c+1) & 0xC0) == 0x80 && str_len < max_num_utf8_byte) {
            continue;
        }

        // now str_buffer should contain a full UTF-8 character
        // c+1 is not a continuation byte, so we read in a full codepoint
        int id = str_lookup(str_buffer, t->sorted_vocab, t->vocab_size);

        if (id != -1) {
            // we found this codepoint in vocab, add it as a token
            tokens[(*n_tokens)++] = id;
        } else {
            // byte_fallback encoding: just encode each byte as a token
            // +3 is here because the first 3 vocab elements are <unk>, <s>, </s>
            // so the individual bytes only start at index 3
            for (int i = 0; i < str_len; i++) {
                tokens[(*n_tokens)++] = (unsigned char)str_buffer[i] + 3;
            }
        }
        str_len = 0; // protect against a sequence of stray UTF8 continuation bytes
    }
}

// ----------------------------------------------------------------------------
// Transformer
// ----------------------------------------------------------------------------

void alloc_run_state(RunState *s, Config config) {
    int kv_dim = config.dim * config.n_kv_heads / config.n_heads;
    checkCudaErrors(hipMallocManaged((void **)&s->x, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->xb, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->xb2, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->hb, config.hidden_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->hb2, config.hidden_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->q, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->key_cache, config.n_layers * config.max_seq_len * kv_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->value_cache, config.n_layers * config.max_seq_len * kv_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->att, config.n_heads * config.max_seq_len * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->logits, config.vocab_size * sizeof(float)));
}

void free_run_state(RunState *s) {
    checkCudaErrors(hipFree((void *)s->x));
    checkCudaErrors(hipFree((void *)s->xb));
    checkCudaErrors(hipFree((void *)s->xb2));
    checkCudaErrors(hipFree((void *)s->hb));
    checkCudaErrors(hipFree((void *)s->hb2));
    checkCudaErrors(hipFree((void *)s->q));
    checkCudaErrors(hipFree((void *)s->key_cache));
    checkCudaErrors(hipFree((void *)s->value_cache));
    checkCudaErrors(hipFree((void *)s->att));
    checkCudaErrors(hipFree((void *)s->logits));
}

void memory_map_weights(TransformerWeights *w, Config config, float *ptr, int shared_weights) {
    int head_size = config.dim / config.n_heads;
    // make sure the multiplications below are done in 64bit to fit the parameter counts of 13B+ models
    unsigned long long n_layers = config.n_layers;
    w->token_embedding_table = ptr;
    ptr += config.vocab_size * config.dim;
    w->rms_att_weight = ptr;
    ptr += n_layers * config.dim;
    w->wq = ptr;
    ptr += n_layers * config.dim * (config.n_heads * head_size);
    w->wk = ptr;
    ptr += n_layers * config.dim * (config.n_kv_heads * head_size);
    w->wv = ptr;
    ptr += n_layers * config.dim * (config.n_kv_heads * head_size);
    w->wo = ptr;
    ptr += n_layers * (config.n_heads * head_size) * config.dim;
    w->rms_ffn_weight = ptr;
    ptr += n_layers * config.dim;
    w->w1 = ptr;
    ptr += n_layers * config.dim * config.hidden_dim;
    w->w2 = ptr;
    ptr += n_layers * config.hidden_dim * config.dim;
    w->w3 = ptr;
    ptr += n_layers * config.dim * config.hidden_dim;
    w->rms_final_weight = ptr;
    ptr += config.dim;
    ptr += config.max_seq_len * head_size / 2; // skip what used to be freq_cis_real (for RoPE)
    ptr += config.max_seq_len * head_size / 2; // skip what used to be freq_cis_imag (for RoPE)
    w->wcls = shared_weights ? w->token_embedding_table : ptr;
}

void read_checkpoint(char *checkpoint, Transformer *transformer) {
    Config *config = &(transformer->config);
    FILE *file = fopen(checkpoint, "rb");   // "rb" for openning binary file
    if (file == NULL) {fprintf(stderr, "Failed to open checkpoint file %s\n", checkpoint); exit(EXIT_FAILURE);}
    // read in the config header
    if (fread(config, sizeof(Config), 1, file) != 1) {
        fprintf(stderr, "Read config from checkpoint %s failed due to an error or EOF\n", checkpoint); exit(EXIT_FAILURE);
    }
    // negative vocab size is hacky way of signaling unshared weights. bit yikes.
    int shared_weights = config->vocab_size > 0 ? 1 : 0;
    config->vocab_size = abs(config->vocab_size);
    // figure out the file size
    fseek(file, 0, SEEK_END); // move file pointer to end of file
    transformer->file_size = ftell(file);
    fclose(file);
    // memory map the Transformer weights into the data pointer
    transformer->fd = open(checkpoint, O_RDONLY);
    if (transformer->fd == -1) { fprintf(stderr, "open checkpoint failed!\n"); exit(EXIT_FAILURE); }
    checkCudaErrors(hipMallocManaged((void **)&transformer->data, transformer->file_size+1));
    float *weights_ptr = transformer->data + sizeof(Config) / sizeof(float);
    memory_map_weights(&transformer->weights, transformer->config, weights_ptr, shared_weights);
    if (transformer->fd != -1) {close(transformer->fd);}
}

void build_transformer(Transformer *transformer, char *checkpoint_path) {
    // read in Config and the Weights from the checkpoint
    read_checkpoint(checkpoint_path, transformer);
    // allocate the RunState buffers
    alloc_run_state(&transformer->state, transformer->config);
}

void free_transformer(Transformer* t) {
    // close the memory mapping
    checkCudaErrors(hipFree(t->data));
    // free the RunState buffers
    free_run_state(&t->state);
}

// ----------------------------------------------------------------------------
// generation loop
// ----------------------------------------------------------------------------

void generate(Transformer *transformer, Tokenizer *tokenizer, Sampler *sampler, char *prompt, int steps) {
    char *empty_prompt = "";
    if (prompt == NULL) {prompt = empty_prompt;}

    // encode the (string) prompt into tokens sequence
    int num_prompt_tokens = 0;
    int *prompt_tokens = NULL;
    checkCudaErrors(hipMallocManaged((void **)&prompt_tokens, sizeof(int) * strlen(prompt)+3)); // +3 for '\0', ?BOS, ?EOS
    encode(tokenizer, prompt, 1, 0, prompt_tokens, &num_prompt_tokens);
}

// long arguments
static struct option long_options[] = {
    {"model", required_argument, NULL, 'm'},
    {"tokenizer", optional_argument, NULL, 'z'},
    {"temperature", optional_argument, NULL, 't'},
    {"topp", optional_argument, NULL, 'p'},
    {"seed", optional_argument, NULL, 's'},
    {"step", optional_argument, NULL, 'n'},
    {"prompt", required_argument, NULL, 'i'},
    {"mode", optional_argument, NULL, 'M'},
    {"system-prompt", optional_argument, NULL, 'y'},
    {"ngl", optional_argument, NULL, 'l'},
    {"stream", no_argument, NULL, 'S'},
    {"help", optional_argument, NULL, 'h'},
};

void help_msg() {
    fprintf(stderr, "Usage: run main <mode_checkpoint> [options]\n");
    fprintf(stderr, "Example: ./main -i \"Tell me a story\"\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -m, --model <string> model checkpoint path\n");
    fprintf(stderr, "  -z, --tokenizer <string> tokenizer path\n");
    fprintf(stderr, "  -t, --temperature <float> temperatutre in [0,inf], default to 1.0\n");
    fprintf(stderr, "  -p, --topp <float> p value in top-p (nucleus) sampling in [0,1] default 0.9\n");
    fprintf(stderr, "  -s, --seed <int> random seed, default time(NULL)\n");
    fprintf(stderr, "  -n, --step <int> number of steps to run for, default 256. 0 = max_seq_len\n");
    fprintf(stderr, "  -i, --prompt <string> input prompt\n");
    fprintf(stderr, "  -M, --mode <string> mode: generate|chat, default: generate\n");
    fprintf(stderr, "  -y, --system_prompt <string> (optional) system prompt in chat mode\n");
    fprintf(stderr, "  -l, --ngl <int> (optional) number of layers offload to CPU\n");
    fprintf(stderr, "  -S, --stream (optional) whether to stream outputs\n");
    fprintf(stderr, "  -y, --system_prompt <string> (optional) system prompt in chat mode\n");
    fprintf(stderr, "  -h, --help print this message\n");
    exit(EXIT_FAILURE);
}

int main(int argc, char *argv[]) {

    // Parameter setup
    char *checkpoint_path = NULL;   // e.g. models/llama2-7b.bin
    char *tokenizer_path = (char *)"tokenizer.bin";
    float temperature = 1.0f;   // higher temperature leads to more creative generations
    float topp = 0.9f;  // nucleas sampling.
    int steps = 256;
    char *prompt = NULL;
    unsigned long long rng_seed = 0;
    char *mode = (char *)"generate";    // generate|chat
    char *system_prompt = NULL;     // optional system prompt used in chat mode
    bool stream = false;
    int layers = -1;    // layers to offload to CPU

    // parse arguments
    int opt = 0;
    while ((opt = getopt_long(argc, argv, "m:z:t:p:s:n:i:M:y:l:Sh",
                    long_options, NULL)) != -1) {
        switch (opt) {
            case 'm':
                checkpoint_path = optarg;
                printf("checkpoint_path: %s\n", checkpoint_path);
                break;
            case 'z':
                tokenizer_path = optarg;
                printf("tokenizer path: %s\n", tokenizer_path);
                break;
            case 't':
                temperature = atoi(optarg);
                printf("temperature is %f\n", temperature);
                break;
            case 'p':
                topp = atoi(optarg);
                printf("topp is %f\n", topp);
                break;
            case 's':
                rng_seed = atoi(optarg);
                printf("rng seed %llu\n", rng_seed);
                break;
            case 'n':
                steps = atoi(optarg);
                printf("step is %d\n", steps);
                break;
            case 'i':
                prompt = optarg;
                break;
            case 'M':
                mode = optarg;
                break;
            case 'y':
                system_prompt = optarg;
                break;
            case 'l':
                layers = atoi(optarg);
                break;
            case 'S':
                stream = true;
                printf("stream is: %d\n", stream);
                break;
            case 'h':
                help_msg();
                break;
            case '?':
                help_msg();
                break;
            default:
                help_msg();
                break;
        }
    }

    // parameter validation/correction
    if (rng_seed <= 0) {rng_seed = (unsigned long long)time(NULL);}
    if (temperature < 0.0) {temperature = 0.0f;}
    if (topp < 0.0 || 1.0 <= topp) {topp = 0.9f;}
    if (steps < 0) {steps = 0;}

    // build Transformer from given model .bin file
    Transformer transformer;
    build_transformer(&transformer, checkpoint_path);
    if (steps == 0 || steps > transformer.config.max_seq_len) {steps = transformer.config.max_seq_len;}

    // build the tokenizer via the tokenizer .bin file
    Tokenizer tokenizer;
    build_tokenizer(&tokenizer, tokenizer_path, transformer.config.vocab_size);

    // build the Sampler
    Sampler sampler;
    build_sampler(&sampler, transformer.config.vocab_size, temperature, topp, rng_seed);

    // run!
    if (strcmp(mode, "generate") == 0) {
        generate(&transformer, &tokenizer, &sampler, prompt, steps);
    } else {
        fprintf(stderr, "unknown mode: %s\n", mode);
        help_msg();
    }

    free_sampler(&sampler);
    free_tokenizer(&tokenizer);
    free_transformer(&transformer);

    return 0;
}
