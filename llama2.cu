#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <stdint.h>
#include <assert.h>

#include "hip/hip_runtime.h"


#ifndef checkCudaErrors
#define checkCudaErrors(err) __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if (hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}
#endif

// ----------------------------------------------------------------------------
// The Byte Pair Encoding (BPE) Tokenizer that translates strings <-> tokens
// ----------------------------------------------------------------------------

typedef struct {
    char *str;
    int id;
} TokenIndex;

typedef struct {
    char **vocab;
    float **vocab_scores;
    TokenIndex *sorted_vocab;
    int vocab_size;
    unsigned int max_token_length;
    unsigned char byte_pieces[512]; // stores all single-byte strings
} Tokenizer;

typedef struct {
    int dim; // transformer dimension
    int hidden_dim; // ffn layer dimension
    int n_layers; // number of transformer layers
    int n_heads; // number of query heads
    int n_kv_heads; // number of k/v heads
    int vocab_size; // vocabulary size, usually 256 (byte-level)
    int max_seq_len; // maximum sequence length to generate
} Config;

typedef struct {
    // token embedding table
    float* token_embedding_table;    // (vocab_size, dim)
    // weights for rmsnorms
    float* rms_att_weight; // (layer, dim) rmsnorm weights
    float* rms_ffn_weight; // (layer, dim)
    // weights for matmuls. note dim == n_heads * head_size
    float* wq; // (layer, dim, n_heads * head_size)
    float* wk; // (layer, dim, n_kv_heads * head_size)
    float* wv; // (layer, dim, n_kv_heads * head_size)
    float* wo; // (layer, n_heads * head_size, dim)
    // weights for ffn
    float* w1; // (layer, hidden_dim, dim)
    float* w2; // (layer, dim, hidden_dim)
    float* w3; // (layer, hidden_dim, dim)
    // final rmsnorm
    float* rms_final_weight; // (dim,)
    // (optional) classifier weights for the logits, on the last layer
    float* wcls;
} TransformerWeights;

// RunState definition
typedef struct {
    float *x; // activation at current time stamp (dim,)
    float *xb; // same, but insize a residual branch (dim,)
    float *xb2; // an additional buffer just for convenience (dim,)
    float *hb; // buffer for hidden dimension in the ffn (hidden_dim,)
    float *hb2; // bufffer for hidden dimension in the ffn (hidden_dim,)
    float *q; // query (dim,)
    float *k; // key (dim,)
    float *v; // value (dim,)
    float *att; // buffer for the scores/attention avlues (n_heads, seq_len)
    float *logits; // output logits
    // kv cache
    float *key_cache; // (layer, seq_len, dim)
    float *value_cache; // (layer , seq_len, dim)
} RunState;

// Transformer definition
typedef struct {
    Config config;
    TransformerWeights weights; // model weights
    RunState state; // buffer required to store intermediate values during forward pass
    int fd; // file descriptor required for memory mapping, explained later TODO
    float *data; // data pointer, TODO
    uint64_t file_size; // size of the model checkpoint file in bytes
} Transformer;

void alloc_run_state(RunState *s, Config config) {
    int kv_dim = config.dim * config.n_kv_heads / config.n_heads;
    checkCudaErrors(hipMallocManaged((void **)&s->x, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->xb, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->xb2, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->hb, config.hidden_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->hb2, config.hidden_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->q, config.dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->key_cache, config.n_layers * config.max_seq_len * kv_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->value_cache, config.n_layers * config.max_seq_len * kv_dim * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->att, config.n_heads * config.max_seq_len * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&s->logits, config.vocab_size * sizeof(float)));
}

void free_run_state(RunState *s) {
    checkCudaErrors(hipFree((void *)s->x));
    checkCudaErrors(hipFree((void *)s->xb));
    checkCudaErrors(hipFree((void *)s->xb2));
    checkCudaErrors(hipFree((void *)s->hb));
    checkCudaErrors(hipFree((void *)s->hb2));
    checkCudaErrors(hipFree((void *)s->q));
    checkCudaErrors(hipFree((void *)s->key_cache));
    checkCudaErrors(hipFree((void *)s->value_cache));
    checkCudaErrors(hipFree((void *)s->att));
    checkCudaErrors(hipFree((void *)s->logits));
}

void build_tokenizer(Tokenizer *t, char *tokenizer_path, int vocab_size) {
    // should've written the vocab_size into the tokenizer file... sigh
    t->vocab_size = vocab_size;
}

void memory_map_weights(TransformerWeights *w, Config config, float *ptr, int shared_weights) {
    int head_size = config.dim / config.n_heads;
    // make sure the multiplications below are done in 64bit to fit the parameter counts of 13B+ models
    unsigned long long n_layers = config.n_layers;
    w->token_embedding_table = ptr;
    ptr += config.vocab_size * config.dim;
    w->rms_att_weight = ptr;
    ptr += n_layers * config.dim;
    w->wq = ptr;
    ptr += n_layers * config.dim * (config.n_heads * head_size);
    w->wk = ptr;
    ptr += n_layers * config.dim * (config.n_kv_heads * head_size);
    w->wv = ptr;
    ptr += n_layers * config.dim * (config.n_kv_heads * head_size);
    w->wo = ptr;
    ptr += n_layers * (config.n_heads * head_size) * config.dim;
    w->rms_ffn_weight = ptr;
    ptr += n_layers * config.dim;
    w->w1 = ptr;
    ptr += n_layers * config.dim * config.hidden_dim;
    w->w2 = ptr;
    ptr += n_layers * config.hidden_dim * config.dim;
    w->w3 = ptr;
    ptr += n_layers * config.dim * config.hidden_dim;
    w->rms_final_weight = ptr;
    ptr += config.dim;
    ptr += config.max_seq_len * head_size / 2; // skip what used to be freq_cis_real (for RoPE)
    ptr += config.max_seq_len * head_size / 2; // skip what used to be freq_cis_imag (for RoPE)
    w->wcls = shared_weights ? w->token_embedding_table : ptr;
}

void read_checkpoint(char *checkpoint, Transformer *transformer) {
    Config *config = &(transformer->config);
    FILE *file = fopen(checkpoint, "rb");   // "rb" for openning binary file
    if (file == NULL) {fprintf(stderr, "Failed to open checkpoint file %s\n", checkpoint); exit(EXIT_FAILURE);}
    // read in the config header
    if (fread(config, sizeof(Config), 1, file) != 1) {
        fprintf(stderr, "Read config from checkpoint %s failed due to an error or EOF\n", checkpoint); exit(EXIT_FAILURE);
    }
    // negative vocab size is hacky way of signaling unshared weights. bit yikes.
    int shared_weights = config->vocab_size > 0 ? 1 : 0;
    config->vocab_size = abs(config->vocab_size);
    // figure out the file size
    fseek(file, 0, SEEK_END); // move file pointer to end of file
    transformer->file_size = ftell(file);
    fclose(file);
    // memory map the Transformer weights into the data pointer
    transformer->fd = open(checkpoint, O_RDONLY);
    if (transformer->fd == -1) { fprintf(stderr, "open checkpoint failed!\n"); exit(EXIT_FAILURE); }
    checkCudaErrors(hipMallocManaged((void **)&transformer->data, transformer->file_size+1));
    float *weights_ptr = transformer->data + sizeof(Config) / sizeof(float);
    memory_map_weights(&transformer->weights, transformer->config, weights_ptr, shared_weights);
}

void build_transformer(Transformer *transformer, char *checkpoint_path) {
    // read in Config and the Weights from the checkpoint
    read_checkpoint(checkpoint_path, transformer);
    // allocate the RunState buffers
    alloc_run_state(&transformer->state, transformer->config);
}

// long arguments
static struct option long_options[] = {
    {"model", required_argument, NULL, 'm'},
    {"tokenizer", optional_argument, NULL, 'z'},
    {"temperature", optional_argument, NULL, 't'},
    {"topp", optional_argument, NULL, 'p'},
    {"seed", optional_argument, NULL, 's'},
    {"step", optional_argument, NULL, 'n'},
    {"prompt", required_argument, NULL, 'i'},
    {"mode", optional_argument, NULL, 'M'},
    {"system-prompt", optional_argument, NULL, 'y'},
    {"ngl", optional_argument, NULL, 'l'},
    {"stream", no_argument, NULL, 'S'},
    {"help", optional_argument, NULL, 'h'},
};

void help_msg() {
    fprintf(stderr, "Usage: run main <mode_checkpoint> [options]\n");
    fprintf(stderr, "Example: ./main -i \"Tell me a story\"\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -m, --model <string> model checkpoint path\n");
    fprintf(stderr, "  -z, --tokenizer <string> tokenizer path\n");
    fprintf(stderr, "  -t, --temperature <float> temperatutre in [0,inf], default to 1.0\n");
    fprintf(stderr, "  -p, --topp <float> p value in top-p (nucleus) sampling in [0,1] default 0.9\n");
    fprintf(stderr, "  -s, --seed <int> random seed, default time(NULL)\n");
    fprintf(stderr, "  -n, --step <int> number of steps to run for, default 256. 0 = max_seq_len\n");
    fprintf(stderr, "  -i, --prompt <string> input prompt\n");
    fprintf(stderr, "  -M, --mode <string> mode: generate|chat, default: generate\n");
    fprintf(stderr, "  -y, --system_prompt <string> (optional) system prompt in chat mode\n");
    fprintf(stderr, "  -l, --ngl <int> (optional) number of layers offload to CPU\n");
    fprintf(stderr, "  -S, --stream (optional) whether to stream outputs\n");
    fprintf(stderr, "  -y, --system_prompt <string> (optional) system prompt in chat mode\n");
    fprintf(stderr, "  -h, --help print this message\n");
    exit(EXIT_FAILURE);
}

int main(int argc, char *argv[]) {

    // Parameter setup
    char *checkpoint_path = NULL;   // e.g. models/llama2-7b.bin
    char *tokenizer_path = (char *)"tokenizer.bin";
    float temperature = 1.0f;   // higher temperature leads to more creative generations
    float topp = 0.9f;  // nucleas sampling.
    int steps = 256;
    char *prompt = NULL;
    unsigned long long rng_seed = 0;
    char *mode = (char *)"generate";    // generate|chat
    char *system_prompt = NULL;     // optional system prompt used in chat mode
    bool stream = false;
    int layers = -1;    // layers to offload to CPU

    // parse arguments
    int opt = 0;
    while ((opt = getopt_long(argc, argv, "m:z:t:p:s:n:i:M:y:l:Sh",
                    long_options, NULL)) != -1) {
        switch (opt) {
            case 'm':
                checkpoint_path = optarg;
                printf("checkpoint_path: %s\n", checkpoint_path);
                break;
            case 'z':
                tokenizer_path = optarg;
                printf("tokenizer path: %s\n", tokenizer_path);
                break;
            case 't':
                temperature = atoi(optarg);
                printf("temperature is %f\n", temperature);
                break;
            case 'p':
                topp = atoi(optarg);
                printf("topp is %f\n", topp);
                break;
            case 's':
                rng_seed = atoi(optarg);
                printf("rng seed %llu\n", rng_seed);
                break;
            case 'n':
                steps = atoi(optarg);
                printf("step is %d\n", steps);
                break;
            case 'i':
                prompt = optarg;
                break;
            case 'M':
                mode = optarg;
                break;
            case 'y':
                system_prompt = optarg;
                break;
            case 'l':
                layers = atoi(optarg);
                break;
            case 'S':
                stream = true;
                printf("stream is: %d\n", stream);
                break;
            case 'h':
                help_msg();
                break;
            case '?':
                help_msg();
                break;
            default:
                help_msg();
                break;
        }
    }

    // parameter validation/correction
    if (rng_seed <= 0) {rng_seed = (unsigned long long)time(NULL);}
    if (temperature < 0.0) {temperature = 0.0f;}
    if (topp < 0.0 || 1.0 <= topp) {topp = 0.9f;}
    if (steps < 0) {steps = 0;}

    // build Transformer from given model .bin file
    Transformer transformer;
    build_transformer(&transformer, checkpoint_path);
    if (steps == 0 || steps > transformer.config.max_seq_len) {steps = transformer.config.max_seq_len;}

    // build the tokenizer via the tokenizer .bin file
    Tokenizer tokenizer; 
    build_tokenizer(&tokenizer, tokenizer_path, transformer.config.vocab_size);

    return 0;
}
